#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define BLOCKSIZE 8
#define RLEN 160
#define CLEN 160

void MatGenerator(double *M, int N1, int N2) {
	int i;
	for (i = 0; i < N1 * N2; i++) {
		M[i] = (double)(rand() % 100);
	}
}

__global__ static void CUDAMatMul1(const double *A, const double *B, double *C){
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int i;
	int N = RLEN * BLOCKSIZE;
	int M = CLEN * BLOCKSIZE;
	double sum = 0;
	for (i = 0; i < N; i++) {
		sum += A[row * N + i] * B[i * M + col];
	}
	C[row * M + col] = sum;
}

__global__ static void CUDAMatMul2(const double *A, const double *B, double *C) {
	int BlockRow = blockIdx.y;
	int BlockCol = blockIdx.x;
	int row = threadIdx.y;
	int col = threadIdx.x;
	int N = RLEN * BLOCKSIZE;
	int M = CLEN * BLOCKSIZE;
	int i, j;
	double sum = 0;
	for (i = 0; i < (N / BLOCKSIZE); i++) {
		__shared__ double Asub[BLOCKSIZE][BLOCKSIZE];
		__shared__ double Bsub[BLOCKSIZE][BLOCKSIZE];
		Asub[row][col] = A[(blockDim.y * blockIdx.y + threadIdx.y) * N + (i * BLOCKSIZE + threadIdx.x)];
		Bsub[row][col] = B[(i * BLOCKSIZE + threadIdx.y) * M + (blockDim.x * blockIdx.x + threadIdx.x)];
		__syncthreads();
		for (j = 0; j < BLOCKSIZE; j++) {
			sum += Asub[row][j] * Bsub[j][col];
		}
		__syncthreads();
	}
	C[(blockDim.y * blockIdx.y + threadIdx.y) * M + (blockDim.x * blockIdx.x + threadIdx.x)] = sum;
}

double execute1(double *A, double *B, double *D) {
	int i, j, k;
	int N = RLEN * BLOCKSIZE;
	int M = CLEN * BLOCKSIZE;
	LARGE_INTEGER nFreq = { 0 };
	LARGE_INTEGER nBeginTime = { 0 };
	LARGE_INTEGER nEndTime = { 0 };
	double totaltime;
	double sum;
	SetThreadAffinityMask(GetCurrentThread(), 1);
	QueryPerformanceFrequency(&nFreq);
	QueryPerformanceCounter(&nBeginTime);
	for (i = 0; i < N; i++) {
		for (j = 0; j < M; j++) {
			sum = 0;
			for (k = 0; k < N; k++) {
				sum += A[i * N + k] * B[k * M + j];
			}
			D[i * M + j] = sum;
		}
	}
	QueryPerformanceCounter(&nEndTime);
	totaltime = (double)(nEndTime.QuadPart - nBeginTime.QuadPart) / ((double)nFreq.QuadPart);
	return totaltime;
}

double execute2(double *A, double *B, double *C) {
	int N = RLEN * BLOCKSIZE;
	int M = CLEN * BLOCKSIZE;
	double *cudaA, *cudaB, *cudaC;
	LARGE_INTEGER nFreq;
	LARGE_INTEGER nBeginTime;
	LARGE_INTEGER nEndTime;
	double totaltime;
	QueryPerformanceFrequency(&nFreq);
	//�����Կ��ڴ�
	hipMalloc((void **)&cudaA, sizeof(double) * N * N);
	hipMalloc((void **)&cudaB, sizeof(double) * N * M);
	hipMalloc((void **)&cudaC, sizeof(double) * N * M);
	//�����ɵľ����Ƶ��Կ��ڴ���
	QueryPerformanceCounter(&nBeginTime);
	hipMemcpy(cudaA, A, sizeof(double) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, B, sizeof(double) * N * M, hipMemcpyHostToDevice);
	dim3 block(BLOCKSIZE, BLOCKSIZE);
	dim3 grid(CLEN, RLEN);
	CUDAMatMul1 << < grid, block >> >(cudaA, cudaB, cudaC);
	hipMemcpy(C, cudaC, sizeof(double) * N * M, hipMemcpyDeviceToHost);
	QueryPerformanceCounter(&nEndTime);
	totaltime = (double)(nEndTime.QuadPart - nBeginTime.QuadPart) / ((double)nFreq.QuadPart);
	hipFree(cudaA);
	hipFree(cudaB);
	hipFree(cudaC);
	return totaltime;
}

double execute3(double *A, double *B, double *C) {
	int N = RLEN * BLOCKSIZE;
	int M = CLEN * BLOCKSIZE;
	double *cudaA, *cudaB, *cudaC;
	LARGE_INTEGER nFreq;
	LARGE_INTEGER nBeginTime;
	LARGE_INTEGER nEndTime;
	double totaltime;
	QueryPerformanceFrequency(&nFreq);
	//�����Կ��ڴ�
	hipMalloc((void **)&cudaA, sizeof(double) * N * N);
	hipMalloc((void **)&cudaB, sizeof(double) * N * M);
	hipMalloc((void **)&cudaC, sizeof(double) * N * M);
	//�����ɵľ����Ƶ��Կ��ڴ���
	QueryPerformanceCounter(&nBeginTime);
	hipMemcpy(cudaA, A, sizeof(double) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(cudaB, B, sizeof(double) * N * M, hipMemcpyHostToDevice);
	dim3 block(BLOCKSIZE, BLOCKSIZE);
	dim3 grid(CLEN, RLEN);
	CUDAMatMul2 << < grid, block >> >(cudaA, cudaB, cudaC);
	hipMemcpy(C, cudaC, sizeof(double) * N * M, hipMemcpyDeviceToHost);
	QueryPerformanceCounter(&nEndTime);
	totaltime = (double)(nEndTime.QuadPart - nBeginTime.QuadPart) / (double)nFreq.QuadPart;
	hipFree(cudaA);
	hipFree(cudaB);
	hipFree(cudaC);
	return totaltime;
}

int main(int argc, char* argv[]){
	double *A, *B, *C1, *C2, *D;
	int i, j, k, M, N;
	int tag1 = 1, tag2 = 1;
	double time1, time2, time3;
	double max_err = 0, ave_err = 0;
	char result[2][10] = { "false!\0", "true!\0" };
	N = RLEN * BLOCKSIZE;
	M = CLEN * BLOCKSIZE;

	//�����ڴ�ռ�
	A = (double *)malloc(N * N * sizeof(double));
	B = (double *)malloc(N * M * sizeof(double));
	C1 = (double *)malloc(N * M * sizeof(double));
	C2 = (double *)malloc(N * M * sizeof(double));
	D = (double *)malloc(N * M * sizeof(double));
	//��ʼ������A�;���B
	srand(0);
	MatGenerator(A, N, N);
	MatGenerator(B, N, M);

	time1 = execute1(A, B, D);
	time2 = execute2(A, B, C1);
	time3 = execute3(A, B, C2);
	printf("Serial Method:\n");
	printf("time cost is %lfs\n", time1);
	printf("CUDA method 1: %s\n", result[tag1]);
	printf("time cost is %lfs.\n", time2);
	printf("CUDA method 2: %s\n", result[tag2]);
	printf("time cost is %lfs.\n", time3);
	system("pause");
	return 0;
}